#include "option_pricer.cuh"
#include "kernels.cuh"

OptimizedCRROptionPricer::OptimizedCRROptionPricer(int batch_size_, int steps_, 
                                                 double tol_, int max_iter_, 
                                                 hipStream_t stream_, bool use_pinned_memory_)
    : batch_size(batch_size_), steps(steps_), tol(tol_), max_iter(max_iter_), 
      stream(stream_), use_pinned_memory(use_pinned_memory_) {
    allocateMemory();
}

OptimizedCRROptionPricer::~OptimizedCRROptionPricer() {
    freeMemory();
}

void OptimizedCRROptionPricer::allocateMemory() {
    size_t size = batch_size * sizeof(double);
    size_t int_size = batch_size * sizeof(int);
    
    // Allocate host memory (pinned if requested)
    if (use_pinned_memory) {
        CHECK_CUDA_ERROR(hipHostMalloc(&h_S, size));
        CHECK_CUDA_ERROR(hipHostMalloc(&h_K, size));
        CHECK_CUDA_ERROR(hipHostMalloc(&h_r, size));
        CHECK_CUDA_ERROR(hipHostMalloc(&h_q, size));
        CHECK_CUDA_ERROR(hipHostMalloc(&h_T, size));
        CHECK_CUDA_ERROR(hipHostMalloc(&h_marketPrices, size));
        CHECK_CUDA_ERROR(hipHostMalloc(&h_optionType, int_size));
    }
    
    // Allocate device memory
    CHECK_CUDA_ERROR(hipMallocAsync(&d_S, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_K, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_r, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_q, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_T, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_optionType, int_size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_marketPrices, size, stream));
    
    // Pricing buffers
    CHECK_CUDA_ERROR(hipMallocAsync(&d_price, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_sigma, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_price_low, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_price_high, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_price_mid, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_ivResults, size, stream));
    
    // Binomial tree workspace
    size_t prices_size = batch_size * (steps + 1) * sizeof(double);
    CHECK_CUDA_ERROR(hipMallocAsync(&d_prices, prices_size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_values, prices_size, stream));
    
    // Greeks buffers
    CHECK_CUDA_ERROR(hipMallocAsync(&d_greeks_delta, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_greeks_gamma, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_greeks_theta, size, stream));
    CHECK_CUDA_ERROR(hipMallocAsync(&d_greeks_vega, size, stream));
}

void OptimizedCRROptionPricer::freeMemory() {
    // Free host memory
    if (use_pinned_memory) {
        hipHostFree(h_S);
        hipHostFree(h_K);
        hipHostFree(h_r);
        hipHostFree(h_q);
        hipHostFree(h_T);
        hipHostFree(h_marketPrices);
        hipHostFree(h_optionType);
    }
    
    // Free device memory asynchronously
    hipFreeAsync(d_S, stream);
    hipFreeAsync(d_K, stream);
    hipFreeAsync(d_r, stream);
    hipFreeAsync(d_q, stream);
    hipFreeAsync(d_T, stream);
    hipFreeAsync(d_optionType, stream);
    hipFreeAsync(d_marketPrices, stream);
    
    hipFreeAsync(d_price, stream);
    hipFreeAsync(d_prices, stream);
    hipFreeAsync(d_values, stream);
    hipFreeAsync(d_sigma, stream);
    hipFreeAsync(d_price_low, stream);
    hipFreeAsync(d_price_high, stream);
    hipFreeAsync(d_price_mid, stream);
    hipFreeAsync(d_ivResults, stream);
    
    hipFreeAsync(d_greeks_delta, stream);
    hipFreeAsync(d_greeks_gamma, stream);
    hipFreeAsync(d_greeks_theta, stream);
    hipFreeAsync(d_greeks_vega, stream);
}

void OptimizedCRROptionPricer::setData(const std::vector<double>& S, const std::vector<double>& K,
                                      const std::vector<double>& r, const std::vector<double>& q,
                                      const std::vector<double>& T, const std::vector<int>& optionType,
                                      const std::vector<double>& marketPrices) {
    
    size_t size = batch_size * sizeof(double);
    size_t int_size = batch_size * sizeof(int);
    
    // Copy to host buffers if using pinned memory
    if (use_pinned_memory) {
        std::copy(S.begin(), S.end(), h_S);
        std::copy(K.begin(), K.end(), h_K);
        std::copy(r.begin(), r.end(), h_r);
        std::copy(q.begin(), q.end(), h_q);
        std::copy(T.begin(), T.end(), h_T);
        std::copy(optionType.begin(), optionType.end(), h_optionType);
        std::copy(marketPrices.begin(), marketPrices.end(), h_marketPrices);
        
        // Copy from pinned host memory to device (faster)
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_S, h_S, size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_K, h_K, size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_r, h_r, size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_q, h_q, size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_T, h_T, size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_optionType, h_optionType, int_size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_marketPrices, h_marketPrices, size, hipMemcpyHostToDevice, stream));
    } else {
        // Copy directly from host vectors to device
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_S, S.data(), size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_K, K.data(), size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_r, r.data(), size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_q, q.data(), size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_T, T.data(), size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_optionType, optionType.data(), int_size, hipMemcpyHostToDevice, stream));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_marketPrices, marketPrices.data(), size, hipMemcpyHostToDevice, stream));
    }
}

void OptimizedCRROptionPricer::computeAllInOne(std::vector<double>& impliedVols, 
                                             std::vector<Greeks>& greeks,
                                             double dSpot, double dVol) {
    // Launch the fused kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (batch_size + threadsPerBlock - 1) / threadsPerBlock;
    
    fusedComputationKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        steps, batch_size, d_marketPrices, d_S, d_K, d_r, d_q, d_T, d_optionType,
        d_ivResults, d_greeks_delta, d_greeks_gamma, d_greeks_theta, d_greeks_vega,
        dSpot, dVol, tol, max_iter);
    
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // Retrieve results
    impliedVols.resize(batch_size);
    CHECK_CUDA_ERROR(hipMemcpyAsync(impliedVols.data(), d_ivResults, 
                                    batch_size * sizeof(double), 
                                    hipMemcpyDeviceToHost, stream));
    
    // Prepare Greeks
    greeks.resize(batch_size);
    
    // Temporary vectors for Greeks results
    std::vector<double> delta(batch_size);
    std::vector<double> gamma(batch_size);
    std::vector<double> theta(batch_size);
    std::vector<double> vega(batch_size);
    
    // Copy Greeks from device to host
    CHECK_CUDA_ERROR(hipMemcpyAsync(delta.data(), d_greeks_delta, 
                                    batch_size * sizeof(double), 
                                    hipMemcpyDeviceToHost, stream));
    CHECK_CUDA_ERROR(hipMemcpyAsync(gamma.data(), d_greeks_gamma, 
                                    batch_size * sizeof(double), 
                                    hipMemcpyDeviceToHost, stream));
    CHECK_CUDA_ERROR(hipMemcpyAsync(theta.data(), d_greeks_theta, 
                                    batch_size * sizeof(double), 
                                    hipMemcpyDeviceToHost, stream));
    CHECK_CUDA_ERROR(hipMemcpyAsync(vega.data(), d_greeks_vega, 
                                    batch_size * sizeof(double), 
                                    hipMemcpyDeviceToHost, stream));
    
    // Wait for all copies to complete
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    
    // Fill Greeks structure
    for (int i = 0; i < batch_size; i++) {
        greeks[i].delta = delta[i];
        greeks[i].gamma = gamma[i];
        greeks[i].theta = theta[i];
        greeks[i].vega = vega[i];
    }
}