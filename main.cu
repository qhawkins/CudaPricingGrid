#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm>
#include <chrono>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <queue>
#include <thread>
#include <mutex>
#include <condition_variable>
#include <functional>
#include <future>

#include "structs.h"
#include "kernels.cuh"
#include "CRROptionPricer.cuh"


class ThreadPool {
public:
    ThreadPool(size_t threads) : stop(false) {
        for(size_t i = 0; i < threads; ++i)
            workers.emplace_back(
                [this]
                {
                    for(;;)
                    {
                        std::packaged_task<void()> task;
                        {
                            std::unique_lock<std::mutex> lock(this->queue_mutex);
                            this->condition.wait(lock,
                                [this]{ return this->stop || !this->tasks.empty(); });
                            if(this->stop && this->tasks.empty())
                                return;
                            task = std::move(this->tasks.front());
                            this->tasks.pop();
                        }
                        task();
                    }
                }
            );
    };

    std::future<std::vector<OptionData>> enqueue(std::function<std::vector<OptionData>()> task) {
        std::packaged_task<std::vector<OptionData>()> packaged_task(std::move(task));
        std::future<std::vector<OptionData>> res = packaged_task.get_future();
        {
            std::unique_lock<std::mutex> lock(queue_mutex);
            if(stop)
                throw std::runtime_error("enqueue on stopped ThreadPool");
            tasks.emplace(std::move(packaged_task));
        }
        condition.notify_one();
        return res;
    };

    ~ThreadPool() {
        {
            std::unique_lock<std::mutex> lock(queue_mutex);
            stop = true;
        }
        condition.notify_all();
        for(std::thread &worker: workers)
            worker.join();
    };

private:
    std::vector<std::thread> workers;
    std::queue<std::packaged_task<void()>> tasks;
    
    std::mutex queue_mutex;
    std::condition_variable condition;
    bool stop;
};

// Function to process a single batch
std::vector<OptionData> processBatch(const std::vector<OptionData>& batch, 
                                     hipStream_t& stream) 
{
    int batch_size = batch.size();
    std::vector<double> S(batch_size);
    std::vector<double> K(batch_size);
    std::vector<double> r(batch_size);
    std::vector<double> q(batch_size, 0.0); // Assuming no dividends
    std::vector<double> T(batch_size);
    std::vector<int> type(batch_size);
    std::vector<double> marketPrices(batch_size);

    std::vector<double> dSpot(batch_size);
    std::vector<double> dStrike(batch_size);
    std::vector<double> dRate(batch_size, 0.0001);
    std::vector<double> dYield(batch_size, 0.0001);
    std::vector<double> dTime(batch_size, 1.0 / 365.0);
    std::vector<double> dVol(batch_size, 0.01);

    for (int i = 0; i < batch_size; i++) {
        S[i] = batch[i].underlying_price;
        K[i] = batch[i].strike_price;
        r[i] = batch[i].rfr;
        T[i] = batch[i].years_to_expiration;
        type[i] = batch[i].option_type;
        marketPrices[i] = batch[i].market_price;
        dSpot[i] = 0.01 * S[i];
        dStrike[i] = 0.01 * K[i];
    }

    // Initialize pricer
    CRROptionPricer pricer(batch_size, marketPrices.data(), S.data(), K.data(), 
                          r.data(), q.data(), T.data(), 1000, type.data(), 
                          1e-5, 1000, stream);
    // Compute implied volatilities
    std::vector<double> impliedVols;
    pricer.computeImpliedVolatility(impliedVols);

    // Define GreekParams
    GreekParams params;
    params.dSpot = dSpot;
    params.dStrike = dStrike;
    params.dRate = dRate;
    params.dYield = dYield;
    params.dTime = dTime;
    params.dVol = dVol;

    // Calculate Greeks
    std::vector<Greeks> greeks = pricer.calculateAllGreeks(params);

    // Assign results to options
    std::vector<OptionData> results;
    results.reserve(batch_size);
    for (int i = 0; i < batch_size; i++) {
        OptionData option = batch[i];
        //option.modelPrice = host_price[i];
        option.impliedVolatility = impliedVols[i];
        if (option.impliedVolatility < 0) {
            // Handle failure to compute IV
            continue;
        }
        // Assign Greeks
        option.delta = greeks[i].delta;
        option.theta = greeks[i].theta;
        option.vega = greeks[i].vega;
        option.rho = greeks[i].rho;

        // Second-order Greeks
        option.gamma = greeks[i].gamma;
        option.vanna = greeks[i].vanna;
        option.charm = greeks[i].charm;
        option.vomma = greeks[i].vomma;
        option.veta = greeks[i].veta;
        option.vera = greeks[i].vera;

        // Third-order Greeks
        option.speed = greeks[i].speed;
        option.zomma = greeks[i].zomma;
        option.color = greeks[i].color;
        option.ultima = greeks[i].ultima;

        results.push_back(option);
    }

    return results;
}

std::vector<OptionData> read_csv(const std::string& filename) {
    std::vector<OptionData> options;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return options;
    }

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        OptionData option;
        std::string token;
        std::vector<std::string> tokens;

        while (std::getline(iss, token, ',')) {
            //skip the first line
            if (token == "Contract") {
                break;
            }
            tokens.push_back(token);
        }

        if (tokens.size() == 8) {
            try {
                option.market_price = std::stod(tokens[0]);
                option.strike_price = std::stod(tokens[1]);
                option.underlying_price = std::stod(tokens[2]);
                option.years_to_expiration = std::stod(tokens[3]);
                //if (option.years_to_expiration <= .01) {
                //    continue;
                //}
                option.rfr = std::stod(tokens[7])/100;
                char contract_type = tokens[4][0];
                option.contract_id = tokens[5];
                option.timestamp = tokens[6];
                option.option_type = (contract_type == 'C' || contract_type == 'c') ? 0 : 1;
                //double itm_perc = (option.option_type == "call") ? (option.underlying_price - option.strike_price) / option.underlying_price : (option.strike_price - option.underlying_price) / option.underlying_price;
                //std::cout << "ITM Percentage: " << itm_perc << std::endl;
                //if (itm_perc < -.05 || itm_perc > 0.05) {
                //    continue;
               // }
                options.push_back(option);
                if (options.size() == 16384) {
                    break;
                }
            } catch (const std::exception& e) {
                std::cerr << "Error parsing line: " << line << " - " << e.what() << std::endl;
            }
        } else {
            std::cerr << "Error parsing line: " << line << " - Incorrect number of fields" << std::endl;
            std::cout << tokens.size();
            exit(1938);
        }
    }

    file.close();
    return options;
}

int main() {
    std::string input_filename = "/home/qhawkins/Desktop/GMEStudy/timed_opra_clean_mc.csv";
    std::string output_filename = "/home/qhawkins/Desktop/GMEStudy/implied_volatilities_mc.csv";
    std::vector<OptionData> options = read_csv(input_filename);

    const int NUM_STREAMS = 32; // Adjust based on your GPU capabilities
    std::vector<hipStream_t> streams(NUM_STREAMS);
    for (int i = 0; i < NUM_STREAMS; ++i) {
        CHECK_CUDA_ERROR(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));
    }

    const int BATCH_SIZE = 512;

    ThreadPool* pool = new ThreadPool(16);

    std::vector<std::future<std::vector<OptionData>>> futures;
    for (size_t i = 0; i < options.size(); i += BATCH_SIZE) {
        size_t end = std::min(i + BATCH_SIZE, options.size());
        //i = i >= end ? end-BATCH_SIZE : i; 
        std::vector<OptionData> batch(options.begin() + i, options.begin() + end);
        //std::cout << "Batch S: " << batch[0].underlying_price << " K: " << batch[0].strike_price << " r: " << batch[0].rfr << " T: " << batch[0].years_to_expiration << std::endl;
        futures.push_back(pool->enqueue([batch, &streams, i, NUM_STREAMS]()->std::vector<OptionData> {
            return processBatch(batch, streams[i / BATCH_SIZE % NUM_STREAMS]);
        }));
    }
    std::cout << "Queue created" << std::endl;
    // Collect results in order
    std::vector<std::vector<OptionData>> results;
    auto start = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < futures.size(); ++i) {
        futures[i].wait();
    }
    for (size_t i = 0; i < futures.size(); ++i) {
        std::cout << "Waiting for future " << i << std::endl;
        results.push_back(futures[i].get());
    }
    std::vector<OptionData> final_results;
    for (const auto& result : results) {
        for (const auto& option : result) {
            final_results.push_back(option);
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    // Clean up CUDA streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
    }
    // Print or save results
    for (const auto& result : final_results) {
        std::cout << "Contract ID: " << result.contract_id << std::endl;
        std::cout << "Timestamp: " << result.timestamp << std::endl;
        std::cout << "Market Price: " << result.market_price << std::endl;
        std::cout << "Implied Volatility: " << result.impliedVolatility << std::endl;
        std::cout << "Delta: " << result.delta << std::endl;
        std::cout << "Gamma: " << result.gamma << std::endl;
        std::cout << "Theta: " << result.theta << std::endl;
        std::cout << "Vega: " << result.vega << std::endl;
        std::cout << "Rho: " << result.rho << std::endl;
        std::cout << "Vanna: " << result.vanna << std::endl;
        std::cout << "Charm: " << result.charm << std::endl;
        std::cout << "Vomma: " << result.vomma << std::endl;
        std::cout << "Veta: " << result.veta << std::endl;
        std::cout << "Vera: " << result.vera << std::endl;
        std::cout << "Speed: " << result.speed << std::endl;
        std::cout << "Zomma: " << result.zomma << std::endl;
        std::cout << "Color: " << result.color << std::endl;
        std::cout << "Ultima: " << result.ultima << std::endl;
        std::cout << std::endl;
    }
    std::cout << "Elapsed time: " << elapsed.count() << "s" << std::endl;


    return 0;
}